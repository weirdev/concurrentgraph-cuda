#include "hip/hip_runtime.h"
// Compressed sparse row format
// Rows transmit to columns
#include <math.h>
#include "hip/hip_runtime.h"
#include "bfs_csr_kernel.h"

__global__ void breadth_first_search_csr_gpu(unsigned int* cum_row_indexes, unsigned int* column_indexes, 
                                        int* matrix_data, unsigned int* in_infections,
                                        unsigned int* out_infections, unsigned int rows) {
    unsigned int row = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < rows) {
        if (in_infections[row] == 1) {
            out_infections[row] = 1;

            unsigned int row_start = cum_row_indexes[row]; 
            unsigned int row_end = cum_row_indexes[row+1];

            for (int i = row_start; i < row_end; i++) {
                int timesteps_to_transmission = matrix_data[i];
                if (timesteps_to_transmission != 0) {
                    if (timesteps_to_transmission == 1) {
                        out_infections[column_indexes[i]] = 1;
                    }
                    matrix_data[i] -= 1;
                }
            }
        }
    }
}

void internal_breadth_first_search_csr_gpu(unsigned int* cum_row_indexes, unsigned int* column_indexes, 
                                        int* matrix_data, unsigned int* in_infections,
                                        unsigned int* out_infections, unsigned int rows) {
    // declare the number of blocks per grid and the number of threads per block
    // use 1 to 512 threads per block
    dim3 threadsPerBlock(rows);
    dim3 blocksPerGrid(1);
    if (rows > 512) {
        threadsPerBlock.x = 512;
        blocksPerGrid.x = ceil(double(rows)/double(threadsPerBlock.x));
    }

    breadth_first_search_csr_gpu<<<blocksPerGrid,threadsPerBlock>>>(cum_row_indexes, 
        column_indexes, matrix_data, in_infections, out_infections, rows);
}
